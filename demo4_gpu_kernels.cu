#include "hip/hip_runtime.h"
/** 
 */

#include "demo4_gpu_thread.h"

extern hipfftHandle g_stPlan1;
extern float* g_pf4FFTIn_d;
extern float* g_pf4FFTOut1_d;

extern hipfftHandle g_stPlan2;
extern float* g_pf4FFTOut2_d;

extern hipfftHandle g_stPlan3;
extern float* g_pf4FFTOut3_d;

extern hipfftHandle g_stPlan4;
extern float* g_pf4FFTOut4_d;

extern hipfftHandle g_stPlan5;
extern float* g_pf4FFTOut5_d;



__global__ void CopyDataForFFT(char *pc4Data,
                               float *pf4FFTIn)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    pf4FFTIn[i] = (float) pc4Data[i];
    return;
}

/* function that performs the FFT - not a kernel, just a wrapper to an
   API call */
int DoFFT()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecR2C(g_stPlan1,
                             (hipfftReal*) g_pf4FFTIn_d,
                             (hipfftComplex*) g_pf4FFTOut1_d);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT1 failed!\n");
        return EXIT_FAILURE;
    }
    
    iCUFFTRet = hipfftExecR2C(g_stPlan2, (hipfftReal*) g_pf4FFTIn_d, (hipfftComplex*) g_pf4FFTOut2_d);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT2 failed!\n");
        return EXIT_FAILURE;
    }
    
    iCUFFTRet = hipfftExecR2C(g_stPlan3, (hipfftReal*) g_pf4FFTIn_d, (hipfftComplex*) g_pf4FFTOut3_d);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT3 failed!\n");
        return EXIT_FAILURE;
    }

    iCUFFTRet = hipfftExecR2C(g_stPlan4, (hipfftReal*) g_pf4FFTIn_d, (hipfftComplex*) g_pf4FFTOut4_d);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT4 failed!\n");
        return EXIT_FAILURE;
    }

    iCUFFTRet = hipfftExecR2C(g_stPlan5, (hipfftReal*) g_pf4FFTIn_d, (hipfftComplex*) g_pf4FFTOut5_d);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT5 failed!\n");
        return EXIT_FAILURE;
    }
    
    return EXIT_SUCCESS;
}

__global__ void Accumulate(float2 *pf4FFTOut, 
                           float *pf4SumStokes)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    float2 f4FFTOut = pf4FFTOut[i];
    float f4SumStokes = pf4SumStokes[i];

    f4SumStokes += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));

    pf4SumStokes[i] = f4SumStokes;

    return;
}
/*
__global__ void BatchAccumulate(float2 *pf4FFTOut, 
                                int numBatch,
                                int sizeBatch,
                                float *sumBatches)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    float2 f4FFTOut;
    float isumBatches = sumBatches[i];
    
    for (int n=0; n < numBatch; n++){
        f4FFTOut = pf4FFTOut[i+n*sizeBatch];
        isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
    }

    sumBatches[i] = isumBatches;

    return;
}*/


__global__ void BatchAccumulate(float2 *pf4FFTOut1, float2 *pf4FFTOut2, float2 *pf4FFTOut3, float2 *pf4FFTOut4, float2 *pf4FFTOut5,
    float *sumBatches1, float *sumBatches2, float *sumBatches3, float *sumBatches4, float *sumBatches5,
    int len_odata)
{
int i = (blockIdx.x * blockDim.x) + threadIdx.x;

float2 f4FFTOut;

float isumBatches = sumBatches1[i];
f4FFTOut = pf4FFTOut1[i];
isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
sumBatches1[i] = isumBatches;

isumBatches = sumBatches2[i];
for (int n=0; n < 2; n++){
    f4FFTOut = pf4FFTOut2[i+n*(len_odata/2+1)];
    isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
}
sumBatches2[i] = isumBatches;

isumBatches = sumBatches3[i];
for (int n=0; n < 4; n++){
    f4FFTOut = pf4FFTOut3[i+n*(len_odata/4+1)];
    isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
}
sumBatches3[i] = isumBatches;

isumBatches = sumBatches4[i];
for (int n=0; n < 8; n++){
    f4FFTOut = pf4FFTOut4[i+n*(len_odata/8+1)];
    isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
}
sumBatches4[i] = isumBatches;

isumBatches = sumBatches5[i];
for (int n=0; n < 16; n++){
    f4FFTOut = pf4FFTOut5[i+n*(len_odata/16+1)];
    isumBatches += sqrtf((f4FFTOut.x * f4FFTOut.x) + (f4FFTOut.y * f4FFTOut.y));
}
sumBatches5[i] = isumBatches;

return;
}


__global__ void FIR(float *FFTIn, 
                    float *FIRFFTIn,
                    int len,
                    int FFTnum)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    float coeffs1[5] = {0.1474, 0.1092, 0.2135, 0.1092, 0.1474};
    float coeffs2[5] = {0.2312, 0.2270, 0.2549, 0.2270, 0.2312};
    float coeffs3[5] = {0.1625, 0.3244, 0.3085, 0.3244, 0.1625};
    float coeffs4[5] = {-0.0699, 0.3342, 0.6658, 0.3342, -0.0699};
    float sum=0.0;
    switch(FFTnum){
        case 1:
            for (int n=0; n<5; n++){
                if (i+n >= len) continue;
                sum += coeffs1[n]*FFTIn[i+n];
            }
        case 2:
            for (int n=0; n<5; n++){
                if (i+n >= len) continue;
                sum += coeffs2[n]*FFTIn[i+n];
            }
            break;
        case 3:
            for (int n=0; n<5; n++){
                if (i+n >= len) continue;
                sum += coeffs3[n]*FFTIn[i+n];
            }
            break;
        case 4:
            for (int n=0; n<5; n++){
                if (i+n >= len) continue;
                sum += coeffs4[n]*FFTIn[i+n];
            }
            break;
    }
    
    
    FIRFFTIn[i] = sum;

    return;
}


