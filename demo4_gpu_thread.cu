#include "hip/hip_runtime.h"
/*demo4_gpu_thread.c
 *
 * Get two numbers from input databuffer, calculate them and write the sum to output databuffer.
 */
#ifdef __cplusplus
extern "C"{
#endif
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <sys/types.h>
#include <unistd.h>
#include "hashpipe.h"
#include "demo4_databuf.h"
#include "demo4_gpu_thread.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* plotting */
extern float* g_pfSumPowX;
extern float* g_pfSumPowY;
extern float* g_pfSumStokesRe;
extern float* g_pfSumStokesIm;
extern float* g_pfFreq;
extern float g_fFSamp;

int g_iIsDataReadDone = FALSE;
char4* g_pc4InBuf = NULL;
char4* g_pc4InBufRead = NULL;
char4* g_pc4Data_d = NULL;              /* raw data starting address */
char4* g_pc4DataRead_d = NULL;          /* raw data read pointer */
int g_iNFFT = DEF_LEN_SPEC;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);
float* g_pf4FFTIn_d = NULL;
float2* g_pf4FFTOut_d = NULL;
hipfftHandle g_stPlan = {0};
float* g_pf4SumStokes = NULL;
float* g_pf4SumStokes_d = NULL;
int g_iIsPFBOn = DEF_PFB_ON;
int g_iNTaps = 1;                       /* 1 if no PFB, NUM_TAPS if PFB */
/* BUG: crash if file size is less than 32MB */
int g_iSizeRead = DEF_LEN_SPEC;//DEF_SIZE_READ;
int g_iNumSubBands = DEF_NUM_SUBBANDS;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};
float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;
static int Init(hashpipe_thread_args_t * args)
//int Init()
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    int iRet = EXIT_SUCCESS;
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iMaxThreadsPerBlock = 0;

    iRet = RegisterSignalHandlers();
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Signal-handler registration failed!\n");
        return EXIT_FAILURE;
    }

    /* since CUDASafeCallWithCleanUp() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }

    /* just use the first device */
    CUDASafeCallWithCleanUp(hipSetDevice(0));

    CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer */
    //CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc4DataRead_d, g_iSizeRead));
    CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc4Data_d, g_iSizeRead));
    g_pc4DataRead_d = g_pc4Data_d;

    /* load data from the first file into memory */
    /*iRet = LoadDataToMem();
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! Loading to memory failed!\n");
        return EXIT_FAILURE;
    }*/

    /* calculate kernel parameters */
    if (g_iNFFT < iMaxThreadsPerBlock)
    {
        g_dimBPFB.x = g_iNFFT;
        g_dimBCopy.x = g_iNFFT;
        g_dimBAccum.x = g_iNFFT;
    }
    else
    {
        g_dimBPFB.x = iMaxThreadsPerBlock;
        g_dimBCopy.x = iMaxThreadsPerBlock;
        g_dimBAccum.x = iMaxThreadsPerBlock;
    }
    g_dimGPFB.x = (g_iNumSubBands * g_iNFFT) / iMaxThreadsPerBlock;
    g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / iMaxThreadsPerBlock;
    g_dimGAccum.x = (g_iNumSubBands * g_iNFFT) / iMaxThreadsPerBlock;

    /*iRet = ReadData();
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Reading data failed!\n");
        return EXIT_FAILURE;
    }*/

    CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf4FFTIn_d,
                                       g_iNumSubBands
                                       * g_iNFFT
                                       * sizeof(float)));
    CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf4FFTOut_d,
                                       g_iNumSubBands
                                       * g_iNFFT
                                       * sizeof(float2)));

    g_pf4SumStokes = (float *) malloc(g_iNumSubBands
                                       * g_iNFFT
                                       * sizeof(float));
    if (NULL == g_pf4SumStokes)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf4SumStokes_d,
                                       g_iNumSubBands
                                       * g_iNFFT
                                       * sizeof(float)));
    CUDASafeCallWithCleanUp(hipMemset(g_pf4SumStokes_d,
                                       '\0',
                                       g_iNumSubBands
                                       * g_iNFFT
                                       * sizeof(float)));

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&g_stPlan,
                              FFTPLAN_RANK,
                              &g_iNFFT,
                              &g_iNFFT,
                              FFTPLAN_ISTRIDE,
                              FFTPLAN_IDIST,
                              &g_iNFFT,
                              FFTPLAN_OSTRIDE,
                              FFTPLAN_ODIST,
                              HIPFFT_R2C,
                              FFTPLAN_BATCH);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        return EXIT_FAILURE;
    }
    /*
    iRet = InitPlot();
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Plotting initialisation failed!\n");
        return EXIT_FAILURE;
    }
*/
    return EXIT_SUCCESS;
}

/* function that frees resources */
void CleanUp()
{
    /* free resources */
    if (g_pc4InBuf != NULL)
    {
        free(g_pc4InBuf);
        g_pc4InBuf = NULL;
    }
    if (g_pc4Data_d != NULL)
    {
        (void) hipFree(g_pc4Data_d);
        g_pc4Data_d = NULL;
    }
    if (g_pf4FFTIn_d != NULL)
    {
        (void) hipFree(g_pf4FFTIn_d);
        g_pf4FFTIn_d = NULL;
    }
    if (g_pf4FFTOut_d != NULL)
    {
        (void) hipFree(g_pf4FFTOut_d);
        g_pf4FFTOut_d = NULL;
    }
    if (g_pf4SumStokes != NULL)
    {
        free(g_pf4SumStokes);
        g_pf4SumStokes = NULL;
    }
    if (g_pf4SumStokes_d != NULL)
    {
        (void) hipFree(g_pf4SumStokes_d);
        g_pf4SumStokes_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    if (g_pfSumPowX != NULL)
    {
        free(g_pfSumPowX);
        g_pfSumPowX = NULL;
    }
    if (g_pfSumPowY != NULL)
    {
        free(g_pfSumPowY);
        g_pfSumPowY = NULL;
    }
    if (g_pfSumStokesRe != NULL)
    {
        free(g_pfSumStokesRe);
        g_pfSumStokesRe = NULL;
    }
    if (g_pfSumStokesIm != NULL)
    {
        free(g_pfSumStokesIm);
        g_pfSumStokesIm = NULL;
    }
    if (g_pfFreq != NULL)
    {
        free(g_pfFreq);
        g_pfFreq = NULL;
    }

    /* TODO: check if open */
    cpgclos();
    return;
}

/*
 * Registers handlers for SIGTERM and CTRL+C
 */
int RegisterSignalHandlers()
{
    struct sigaction stSigHandler = {{0}};
    int iRet = EXIT_SUCCESS;

    /* register the CTRL+C-handling function */
    stSigHandler.sa_handler = HandleStopSignals;
    iRet = sigaction(SIGINT, &stSigHandler, NULL);
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Handler registration failed for signal %d!\n",
                       SIGINT);
        return EXIT_FAILURE;
    }

    /* register the SIGTERM-handling function */
    stSigHandler.sa_handler = HandleStopSignals;
    iRet = sigaction(SIGTERM, &stSigHandler, NULL);
    if (iRet != EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Handler registration failed for signal %d!\n",
                       SIGTERM);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

/*
 * Catches SIGTERM and CTRL+C and cleans up before exiting
 */
void HandleStopSignals(int iSigNo)
{
    /* clean up */
    CleanUp();

    /* exit */
    exit(EXIT_SUCCESS);

    /* never reached */
    return;
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pCleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}
/*
 * Prints usage information
 */
void PrintUsage(const char *pcProgName)
{
    (void) printf("Usage: %s [options] <data-file>\n",
                  pcProgName);
    (void) printf("    -h  --help                           ");
    (void) printf("Display this usage information\n");
    (void) printf("    -n  --nfft <value>                   ");
    (void) printf("Number of points in FFT\n");
    (void) printf("    -p  --pfb                            ");
    (void) printf("Enable PFB\n");
    (void) printf("    -a  --nacc <value>                   ");
    (void) printf("Number of spectra to add\n");
    (void) printf("    -s  --fsamp <value>                  ");
    (void) printf("Sampling frequency\n");

    return;
}


static void *run(hashpipe_thread_args_t * args)
{
    // Local aliases to shorten access to args fields
    demo4_input_databuf_t *db_in = (demo4_input_databuf_t *)args->ibuf;
    demo4_output_databuf_t *db_out = (demo4_output_databuf_t *)args->obuf;
    hashpipe_status_t st = args->st;
    const char * status_key = args->thread_desc->skey;

    int rv;
    uint64_t mcnt=0;
    int curblock_in=0;
    int curblock_out=0;
    
    int nhits = 0;
    char *data_raw; // raw data will be feed to gpu thread
    data_raw = (char *)malloc(g_iSizeRead*sizeof(char));

    int n_frames; // number of frames has been processed

    int iRet = EXIT_SUCCESS;
    int iSpecCount = 0;
    int iNumAcc = DEF_ACC;
    if(iNumAcc > g_iSizeRead/g_iNFFT){iNumAcc=g_iSizeRead/g_iNFFT;} // if accumulation number larger than data buffer, setit to number spectra frames of buffer
	int n_spec = 0; // number of spectrum
    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    struct timeval stStart = {0};
    struct timeval stStop = {0};
    const char *pcProgName = NULL;
    int iNextOpt = 0;
    /* valid short options */
    const char* const pcOptsShort = "hb:n:pa:s:";
    /* valid long options */
    const struct option stOptsLong[] = {
        { "help",           0, NULL, 'h' },
        { "nsub",           1, NULL, 'b' },
        { "nfft",           1, NULL, 'n' },
        { "pfb",            0, NULL, 'p' },
        { "nacc",           1, NULL, 'a' },
        { "fsamp",          1, NULL, 's' },
        { NULL,             0, NULL, 0   }
    };
	// initialize 
	/*iRet=Init();
    if (iRet != EXIT_SUCCESS)
    {   
        (void) fprintf(stderr, "ERROR! Init failed!\n");
        CleanUp();
    }*/
    while (run_threads()) {

        hashpipe_status_lock_safe(&st);
        hputi4(st.buf, "GPUBLKIN", curblock_in);
        hputs(st.buf, status_key, "waiting");
       	hputi4(st.buf, "GPUBKOUT", curblock_out);
		hputi8(st.buf,"GPUMCNT",mcnt);
        hashpipe_status_unlock_safe(&st);
		n_spec = 0;
        // Wait for new input block to be filled
        while ((rv=demo4_input_databuf_wait_filled(db_in, curblock_in)) != HASHPIPE_OK) {
            if (rv==HASHPIPE_TIMEOUT) {
                hashpipe_status_lock_safe(&st);
                hputs(st.buf, status_key, "blocked");
                hashpipe_status_unlock_safe(&st);
                continue;
            } else {
                hashpipe_error(__FUNCTION__, "error waiting for filled databuf");
                pthread_exit(NULL);
                break;
            }
        }

        // Get a new data block, update status and determine how to handle it
        /*hashpipe_status_lock_safe(&st);
        hputu8(st.buf, "GPUMCNT", db_in->block[curblock_in].header.mcnt);
        hashpipe_status_unlock_safe(&st);*/

        // Wait for new output block to be free
        while ((rv=demo4_output_databuf_wait_free(db_out, curblock_out)) != HASHPIPE_OK) {
            if (rv==HASHPIPE_TIMEOUT) {
                hashpipe_status_lock_safe(&st);
                hputs(st.buf, status_key, "blocked gpu out");
                hashpipe_status_unlock_safe(&st);
                continue;
            } else {
                hashpipe_error(__FUNCTION__, "error waiting for free databuf");
                pthread_exit(NULL);
                break;
            }
        }

        // Note processing status
        hashpipe_status_lock_safe(&st);
        hputs(st.buf, status_key, "processing gpu");
        hashpipe_status_unlock_safe(&st);
	
		//get data from input databuf to local
		memcpy(data_raw,db_in->block[curblock_in].data_block,g_iSizeRead*sizeof(char));
		for(n_frames=0;n_frames < SIZEOF_INPUT_DATA_BUF/g_iSizeRead;n_frames++){
			// write new data to the gpu buffer
			CUDASafeCallWithCleanUp(hipMemcpy(g_pc4Data_d,
				                           data_raw,
				                           g_iSizeRead*sizeof(char),
				                           hipMemcpyHostToDevice));
			/* whenever there is a read, reset the read pointer to the beginning */
			g_pc4DataRead_d = g_pc4Data_d;
            //printf("SIZEOF_INPUT_DATA_BUF/g_iSizeRead is: %d\n",SIZEOF_INPUT_DATA_BUF/g_iSizeRead);
            
            //printf("iNumAcc: %d\n", iNumAcc);
			while(iSpecCount < iNumAcc){
				
                CopyDataForFFT<<<g_dimGCopy, g_dimBCopy>>>(g_pc4DataRead_d,
                                                            g_pf4FFTIn_d);
                CUDASafeCallWithCleanUp(hipDeviceSynchronize());
                iCUDARet = hipGetLastError();
                if (iCUDARet != hipSuccess)
                {
                    (void) fprintf(stderr,
                                    "ERROR: File <%s>, Line %d: %s\n",
                                    __FILE__,
                                    __LINE__,
                                    hipGetErrorString(iCUDARet));
                    /* free resources */
                    CleanUp();
                    //return EXIT_FAILURE;
                }
                /* update the data read pointer */
                g_pc4DataRead_d += (g_iNumSubBands * g_iNFFT);
				

				/* do fft */
				        //printf("do fft...,");
				iRet = DoFFT();
				if (iRet != EXIT_SUCCESS)
				{
					(void) fprintf(stderr, "ERROR! FFT failed!\n");
					CleanUp();
					//return EXIT_FAILURE;
				}
				        //printf("FFT done!\n");

				/* accumulate power x, power y, stokes, if the blanking bit is
				   not set */
				//        printf("do stokes calculation and accumulation...,");
				Accumulate<<<g_dimGAccum, g_dimBAccum>>>(g_pf4FFTOut_d,
					                                     g_pf4SumStokes_d);
				CUDASafeCallWithCleanUp(hipDeviceSynchronize());
				iCUDARet = hipGetLastError();
				if (iCUDARet != hipSuccess)
				{
					(void) fprintf(stderr,
					               "ERROR: File <%s>, Line %d: %s\n",
					               __FILE__,
					               __LINE__,
					               hipGetErrorString(iCUDARet));
					/* free resources */
					CleanUp();
					//return EXIT_FAILURE;
				}
				//        printf("done!\n");
				++iSpecCount;
			}
			if (iSpecCount == iNumAcc)
			{
				//n_spec ++; //bug, starts writing too late, moved to after write
				/* dump to buffer */
				    //printf("copy accumulation data from gpu to cpu memory...,");
				CUDASafeCallWithCleanUp(hipMemcpy(g_pf4SumStokes,
				                                   g_pf4SumStokes_d,
				                                   (g_iNumSubBands
				                                   * g_iNFFT
				                                    * sizeof(float)),
				                                    hipMemcpyDeviceToHost));

				memcpy(db_out->block[curblock_out].Stokes_Full+N_CHANS_PER_SPEC*n_spec,g_pf4SumStokes,N_CHANS_PER_SPEC*sizeof(float));
                    //printf("Stokes to output done!\n");
                n_spec ++; 
                //for (int n=0; n<50; n++){
                 //   printf("fftout[%d]: %d\n",n,g_pf4FFTOut_d[n]);
               // }

				/* NOTE: Plot() will modify data! */
				//    printf("number %d of plot.\n",n_frames);
				//Plot();
				// (void) usleep(5000);

				/* reset time */
				iSpecCount = 0;
				/* zero accumulators */
				CUDASafeCallWithCleanUp(hipMemset(g_pf4SumStokes_d,
				                               '\0',
				                               (g_iNumSubBands
				                                * g_iNFFT
				                                * sizeof(float))));

				/* if time to read from input buffer */
				iProcData = 0;
				(void) gettimeofday(&stStop, NULL);
				/*(void) printf("Time taken (barring Init()): %gs\n",
					  ((stStop.tv_sec + (stStop.tv_usec * USEC2SEC))
					   - (stStart.tv_sec + (stStart.tv_usec * USEC2SEC))));*/

				//return EXIT_SUCCESS;

				//display number of frames in status
				hashpipe_status_lock_safe(&st);
				hputi4(st.buf,"NFRAMES",n_frames);
				hashpipe_status_unlock_safe(&st);
			}
		}
		// Mark output block as full and advance
		demo4_output_databuf_set_filled(db_out, curblock_out);
		curblock_out = (curblock_out + 1) % db_out->header.n_block;

		// Mark input block as free and advance
		demo4_input_databuf_set_free(db_in, curblock_in);
		curblock_in = (curblock_in + 1) % db_in->header.n_block;
		mcnt++;
		/* Check for cancel */
		pthread_testcancel();
	}
	CleanUp();
}

static hashpipe_thread_desc_t demo4_gpu_thread = {
    name: "demo4_gpu_thread",
    skey: "GPUSTAT",
    init: Init,
    //init: NULL,
    run:  run,
    ibuf_desc: {demo4_input_databuf_create},
    obuf_desc: {demo4_output_databuf_create}
};

static __attribute__((constructor)) void ctor()
{
	register_hashpipe_thread(&demo4_gpu_thread);
}
#ifdef __cplusplus
}
#endif
